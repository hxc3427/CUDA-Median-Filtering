#include "hip/hip_runtime.h"
//
//  GPU.cpp
//  hpalab5
//
//  Created by Harshdeep Singh Chawla on 10/11/16.
//  Copyright � 2016 Harshdeep Singh Chawla. All rights reserved.
//


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include "MedianFilter.h"
#include "Bitmap.h"

///Kernelk function
__global__ void MFKernel(unsigned char *inputImage, unsigned char *outputImage, int Width, int Height)
{
	// indexing for thread.
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	//filter mask
	unsigned char filter[9];

	/////checking boundry conditions
	if((idy==0) || (idx==0) || (idy==Height-1) || (idx==Width-1))
				outputImage[idy*Width+idx] = 0;
	else {
		for (int x = 0; x < WINDOW_SIZE; x++) { 
			for (int y = 0; y < WINDOW_SIZE; y++){
				filter[x*WINDOW_SIZE+y] = inputImage[(idy+x-1)*Width+(idx+y-1)];   // setup the filterign window.
			}
		}
		////Sorting in filter
		for (int i = 0; i < 9; i++) {
			for (int j = i + 1; j < 9; j++) {
				if (filter[i] > filter[j]) { 
					//Swap the variables.
					unsigned char tmp = filter[i];
					filter[i] = filter[j];
					filter[j] = tmp;
				}
			}
		}
		outputImage[idy*Width+idx] = filter[4];   //Set output variables.
	}
}

__global__ void MFSharedKernel(unsigned char *inputImage, unsigned char *outputImage, int Width, int Height)
{
	//Set the row and col value for each thread.
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int TILE_SIZE = 16;
	__shared__ unsigned char sharedmem[(TILE_SIZE+2)]  [(TILE_SIZE+2)];  //initialize shared memory

	//Initialize with zero
	if(threadIdx.x == 0)
		sharedmem[threadIdx.x][threadIdx.y+1] = 0;
	else if(threadIdx.x == TILE_SIZE-1)
		sharedmem[threadIdx.x + 2][threadIdx.y+1]=0;
	if (threadIdx.y == 0){
		sharedmem[threadIdx.x+1][threadIdx.y] = 0;
		if(threadIdx.x == 0)
			sharedmem[threadIdx.x][threadIdx.y] = 0;
		else if(threadIdx.x == TILE_SIZE-1)
			sharedmem[threadIdx.x+2][threadIdx.y] = 0;
	}
	else if (threadIdx.y == TILE_SIZE-1){
		sharedmem[threadIdx.x+1][threadIdx.y+2] = 0;
		if(threadIdx.x == TILE_SIZE-1)
			sharedmem[threadIdx.x+2][threadIdx.y+2] = 0;
		else if(threadIdx.x == 0)
			sharedmem[threadIdx.x][threadIdx.y+2] = 0;
	}

	//Setup pixel values
	sharedmem[threadIdx.x+1][threadIdx.y+1] = inputImage[idy*Width+idx];
	//Check for boundry conditions.
	if(threadIdx.x == 0 && (idx>0))
		sharedmem[threadIdx.x][threadIdx.y+1] = inputImage[idy*Width+(idx-1)];
	else if(threadIdx.x == TILE_SIZE-1 && (idx<Width-1))

		sharedmem[threadIdx.x + 2][threadIdx.y+1]= inputImage[idy*Width+(idx+1)];
	if (threadIdx.y == 0 && (idy>0)){
		sharedmem[threadIdx.x+1][threadIdx.y] =inputImage[(idy-1)*Width+idx];

		if(threadIdx.x == 0)
			sharedmem[threadIdx.x][threadIdx.y] = inputImage[(idy-1)*Width+(idx-1)];
		else if(threadIdx.x == TILE_SIZE-1 )
			sharedmem[threadIdx.x+2][threadIdx.y] = inputImage[(idy-1)*Width+(idx+1)];
	}
	else if (threadIdx.y == 0 && (idy<Height-1)){
		sharedmem[threadIdx.x+1][threadIdx.y+2] = inputImage[(idy+1)*Width + idx];
		if(threadIdx.x == TILE_SIZE-1)
			sharedmem[threadIdx.x+2][threadIdx.y+2] =inputImage[(idy+1)*Width+(idx+1)];
		else if(threadIdx.x == 0)
			sharedmem[threadIdx.x][threadIdx.y+2] = inputImage[(idy+1)*Width+(idx-1)];
	}

//	hipDeviceSynchronize();   //Wait for all threads to be done.

	//Setup the filter.
	unsigned char filterVector[9] = {sharedmem[threadIdx.x][threadIdx.y], sharedmem[threadIdx.x+1][threadIdx.y], sharedmem[threadIdx.x+2][threadIdx.y],
                   sharedmem[threadIdx.x][threadIdx.y+1], sharedmem[threadIdx.x+1][threadIdx.y+1], sharedmem[threadIdx.x+2][threadIdx.y+1],
                   sharedmem[threadIdx.x] [threadIdx.y+2], sharedmem[threadIdx.x+1][threadIdx.y+2], sharedmem[threadIdx.x+2][threadIdx.y+2]};

	
	{
		for (int i = 0; i < 9; i++) {
        for (int j = i + 1; j < 9; j++) {
            if (filterVector[i] > filterVector[j]) { 
				//Swap Values.
                char tmp = filterVector[i];
                filterVector[i] = filterVector[j];
                filterVector[j] = tmp;
            }
        }
    }
	outputImage[idy*Width+idx] = filterVector[4];   //Set the output image values.
	}
}



///GPU Function
bool MedianFilterGPU( Bitmap* image, Bitmap* outputImage, bool sharedMemoryUse ){

	//Cuda error and image values.
	hipError_t status;
	int w = image->Width();
	int h = image->Height();

	int bytes =  w * h * sizeof(unsigned char);
	//initialize images.
	unsigned char *inputimage_d;
	hipMalloc((void**) &inputimage_d, bytes);
	hipMemcpy(inputimage_d, image->image, bytes, hipMemcpyHostToDevice);
	
	unsigned char *outputImage_d;
	hipMalloc((void**) &outputImage_d, bytes);
	//take block and grids.
	int TILE_SIZE=16;
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);
	dim3 dimGrid((int)ceil((float)image->Width() / (float)TILE_SIZE),
				(int)ceil((float)image->Height() / (float)TILE_SIZE));

	//Check condition for shared memorey
	if (sharedMemoryUse== false){
	//kernel call
		MFKernel<<<dimGrid, dimBlock>>>(inputimage_d, outputImage_d, w, h);
		hipDeviceSynchronize();
	}
	else{
		MFSharedKernel<<<dimGrid, dimBlock>>>(inputimage_d, outputImage_d, w, h);
		hipDeviceSynchronize();
	}
	
	// save output image to host.
	hipMemcpy(outputImage->image, outputImage_d, bytes, hipMemcpyDeviceToHost);
	status = hipGetLastError();              
	if (status != hipSuccess) {                     
		std::cout << "Kernel failed for hipMemcpy hipMemcpyDeviceToHost: " << hipGetErrorString(status) << 
		std::endl;
		hipFree(inputimage_d);
		hipFree(outputImage_d);
		return false;
	}
	//Free the memory
	hipFree(inputimage_d);
	hipFree(outputImage_d);
	return true;
}